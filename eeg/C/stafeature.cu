#include "hip/hip_runtime.h"
#include "eeg.h"

// Based on code by: Mohammad Tahghighi
int32_t abssum(int np, int32_t *x)
{
    int i;
    int32_t s = 0;

    for (i = 0; i < np; i++) {
        s += abs(x[i]);
    }

    return s;
}

float average(int np, int32_t *x)
{
    int i;
    int32_t s = 0;

    for (i = 0; i < np; i++) {
        s += x[i];
    }

    return ((float) s) / ((float) np);
}

__global__
void gpu_average(int32_t *x, int32_t *blocksums)
{
    //Note that his kernel is merely an example, and is not necessarily the optimal way to calculate the sum/average on a GPU device

    // The id of this thread within our block
    unsigned int threadId = threadIdx.x;

    // The global id if this thread.
    // Since we launch np threads in total, each id maps to one unique element in x
    unsigned int globalId = blockIdx.x*blockDim.x + threadIdx.x;

    // NOTE: for debugging you can print directly from the GPU device
    // however, if you print a lot, or the GPU encounters a serious error, this might fail
    // also performance is horrible, so make sure to disable it for benchmarking
    //printf("Hello from global thread %d, with local id %d\n",globalId,threadId);

    // Lets first copy the data from global GPU memory to shared memory
    // Shared memory is only accesible within a threadblock, but it is much faster to access than global memory once you have data in there
    // Note that by having the keyword "extern" and empty brackets [], the size of the array will be determined at runtime.
    // You will however have to pass the size in bytes as a 3rd argument to the kernel call (see the "sharedMemBytes" variable)
    // If you statically know the size of the shared memory array, it is probably faster to use that (Disclaimer: I did not test this claim)
    extern __shared__ int32_t blockData[];
    blockData[threadId]=x[globalId];

    // We synchronize the threads here, to make sure every thread has copied valid data from global to local memory
    // Otherwise we potentially risk accessing uninitialized data in the shared memory
    __syncthreads();

    // The next step is summation of the elements in our blockData
    // The summation is done in a tree like fashion, as illustrated below
    // 0 1 2 3 4 5 6 7  (number of parallel summations)
    // |/  |/  |/  |/   (4)
    // 1   5   9   13
    // |__/    |__/     (2)
    // 6       22
    // |______/         (1)
    // 28
    for(unsigned int s=1;s<blockDim.x;s*=2){
        // Because the amount of work reduces, we use the threadId to determine which threads get to execute the summation
        // The other threads will idle in the meantime (They will be masked during the execution of the conditional-part)
        if (threadId % (2*s) == 0 ){
            blockData[threadId] += blockData[threadId+s];
        }

        // For each layer of the tree, we have to make sure all threads finish their computations
        // otherwise we could read unsummed results
        __syncthreads();
    }

    //we let 1 selected thread per block write out our local sum to the global memory
    if(threadId==0){

        #ifdef DEBUG
        //example debugging, print the partial sum of each block with the block id
        printf("GPU Block %d sum: %d\n",blockIdx.x, blockData[0]);
        #endif

        //write the sum of this block to the blocksums array
        blocksums[blockIdx.x]=blockData[0];
    }

    //this will return the control to the CPU once all threads finish (reach this point)
    return;
}

float variance(int np, int32_t *x, float avg)
{
    int i;
    float s = 0;

    // Variance = Sum((x - avg)^2)
    for (i = 0; i < np; i++) {
        float tmp = x[i] - avg;
        s += (tmp * tmp);
    }

    return s / ((float) np);
}

float stddev(int np, int32_t *x, float avg)
{
    // Stddev = sqrt(variance)
    float var = variance(np, x, avg);
    return sqrt(var);
}

int mean_crosstimes(int np, int32_t *x, float avg)
{
    int i;
    bool negative = x[0] < avg;
    int count = 0;

    // Count number of zero crossings for (x - avg)
    for (i = 0; i < np; i++) {
        if (negative) {
            if (x[i] > avg) {
                negative = false;
                count++;
            }
        } else {
            if (x[i] < avg) {
                negative = true;
                count++;
            }
        }
    }

    return count;
}

void stafeature(int np, int32_t *x, float *sta)
{
    // Returns sta = [mean, std, abssum, mean_crosstimes)


    #ifdef CPU_ONLY
    //original CPU code
    float avg = average(np, x);
    sta[0] = avg;
    sta[1] = stddev(np, x, avg);
    sta[2] = abssum(np, x);
    sta[3] = mean_crosstimes(np, x, avg);
    #else

    //GPU code

    /*
     *  Our high level strategy to calculate the average in parallel in this example is to split the input array into into a number of blocks (numBlocks).
     *  Each block contains thus np/numBlocks elements
     *  These blocks will be mapped to the Streaming Multiprocessors of the GPU.
     *  For each block we calculate the sum
     *  Finally the sums of all the blocks are added on the CPU
    */

    //NOTE: take care np is a multiple of numBlocks for this example.
    int numBlocks=4;
    int threadsPerBlock=np/numBlocks; //i.e., this should have remainder==0

    //Ignore the next bit untill you are inspecting the gpu kernel code, then refer back to it. On the first read just ignore it ;)
    //Because in this setup the amount of required shared memory depends on np we assume it is only known at runtime. (although you can of course get it from the input and assume it fixed for this assigment)
    //the number of required shared memory bytes need to be passed as a 3rd argument to the kernel call later on
    //see the remarks in the gpu_average code
    int sharedMemBytes = threadsPerBlock*sizeof(int32_t);

    //variable for holding return values of cuda functions
    hipError_t err;

    //start by allocating room for array "x" on the global memory of the GPU
    int32_t* device_x;
    err=hipMalloc(&device_x, np*sizeof(int32_t));

    //Here we check for errors of this cuda call
    //See eeg.h for the implementation of this error check (it's not a default cuda function)
    cudaCheckError(err);

    //also allocate room for the all the sums of the blocks
    int32_t* device_blocksums;
    //Note that room is allocated in global memory for the sum of *each* threadblock
    err=hipMalloc(&device_blocksums, numBlocks*sizeof(int32_t));
    cudaCheckError(err);

    //Now copy array "x" from the CPU to the GPU
    err=hipMemcpy(device_x,x, np*sizeof(int32_t), hipMemcpyHostToDevice);
    cudaCheckError(err);

    //Compute the average on the GPU
    gpu_average<<<numBlocks,threadsPerBlock, sharedMemBytes>>>(device_x, device_blocksums);

    //We use "peekatlasterror" since a kernel launch does not return a hipError_t to check for errors
    cudaCheckError(hipPeekAtLastError());

    //copy the sums of each block back from GPU global memory to CPU memory
    int32_t blocksums[numBlocks];
    err=hipMemcpy(blocksums, device_blocksums, numBlocks*sizeof(int32_t), hipMemcpyDeviceToHost);
    cudaCheckError(err);

    //free the memory on the GPU
    //Optimalisation Hint: if you do not free the memory, the values will be preserved between multiple kernel calls!
    //For example, the x-array will remain in the GPU global memory if you also map other features to the GPU
    err=hipFree(device_x);
    cudaCheckError(err);
    err=hipFree(device_blocksums);
    cudaCheckError(err);

    #ifdef DEBUG
    //print all the block sums calculated by CPU
    for(int b=0;b<numBlocks;b++){
        int sum=0;
        for (int i=0;i<threadsPerBlock;i++)
            sum+=x[b*threadsPerBlock+i];
        printf("CPU Block %d sum: %d\n",b,sum);
    }
    #endif

    //Now add all the block sums on the CPU
    //(Note: if you have many blocks, you might consider mapping this to another GPU call of course)
    int32_t sum=0;
    for(uint32_t blk=0;blk<numBlocks;blk++)
        sum+=blocksums[blk];
    float avg = (float)(sum)/(float)(np);


    #ifdef DEBUG
    //Compare total sum of GPU and CPU
    printf("GPU Total sum: %d\n",sum);
    int cpu_sum=0;
    for(int i=0;i<np;i++)
        cpu_sum+=x[i];
    printf("CPU Total sum: %d\n",cpu_sum);
    #endif

    #ifdef DEBUG
    //compare the average
    printf("GPU average: %f\n",avg);
    printf("CPU average: %f\n",(float)cpu_sum/(float)np);
    #endif

    //calculate all other features on the CPU for this example
    sta[0] = avg;
    sta[1] = stddev(np, x, avg);
    sta[2] = abssum(np, x);
    sta[3] = mean_crosstimes(np, x, avg);

    #endif
}
